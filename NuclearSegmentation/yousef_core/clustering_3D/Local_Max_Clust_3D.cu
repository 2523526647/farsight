#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void InitialClusteringKernel (float* im_vals, unsigned short* local_max_vals, unsigned short* max_response_r, unsigned short* max_response_c, unsigned short* max_response_z , int r, int c, int z, int scale_xy, int scale_z, int offset)
{
	int iGID = blockIdx.x * blockDim.x + threadIdx.x + offset; //global index
	
	int rem = ((long)iGID) % (r*c);
	int k1 = ((int)iGID-rem) / (r*c); 
	int j1 = ((long)rem) % c;
	int i1 = (rem-j1)/c;

	int min_r = (int) max((double)(0.0),(double)(i1-scale_xy));
	int min_c = (int) max((double)(0.0),(double)(j1-scale_xy));
	int min_z = (int) max((double)(0.0),(double)(k1-scale_z));
	int max_r = (int) min((double)(r-1),(double)(i1+scale_xy));
	int max_c = (int) min((double)(c-1),(double)(j1+scale_xy));                         
	int max_z = (int) min((double)(z-1),(double)(k1+scale_z));

	if(local_max_vals[(k1*r*c)+(i1*c)+j1] == 0) //if current pixel is not a seed point			
	{
		float mx = im_vals[(min_z*r*c)+(min_r*c)+min_c];//A[r1][c1][z1];
		
		max_response_r[i1 * (c * z) + j1 * z + k1] = min_r;
		max_response_r[i1 * (c * z) + j1 * z + k1] = min_c;
		max_response_r[i1 * (c * z) + j1 * z + k1] = min_z;
	    
		for(int i= min_r; i<= max_r; i++)
		{
			for(int j= min_c; j <= max_c; j++)
			{
				for(int k = min_z; k <= max_z; k++)
				{
					if(im_vals[(k*r*c)+(i*c)+j] >= mx)
					{
						mx = im_vals[(k*r*c)+(i*c)+j];//A[i][j][k];

						max_response_r[i1 * (c * z) + j1 * z + k1] = i;
						max_response_c[i1 * (c * z) + j1 * z + k1] = j;
						max_response_z[i1 * (c * z) + j1 * z + k1] = k;
					}
				}
			}
		}                                          
	}
}


extern "C"
void initialClustering_CUDA (float* im_vals, unsigned short* local_max_vals, unsigned short* max_response_r, unsigned short* max_response_c, unsigned short* max_response_z , int r, int c, int z, int scale_xy, int scale_z)
{
	cout << "Entering initialClustering_CUDA" << endl;
	
	hipError_t errorcode;

	float* dev_im_vals; 
	unsigned short* dev_local_max_vals;
	unsigned short* dev_max_response_r;
	unsigned short* dev_max_response_c;
	unsigned short* dev_max_response_z;

	size_t free_mem, total_mem;
	hipMemGetInfo(&free_mem, &total_mem);

	cout << free_mem / (double)(1024 * 1024) << " " << total_mem / (double)(1024 * 1024) << endl;

	cout << "Allocating " << (sizeof(*im_vals) * r * c * z)/(double)(1024*1024) << " MB of memory on GPU for im_vals" << endl;
	cout << "Allocating " << (sizeof(*local_max_vals) * r * c * z)/(double)(1024*1024) << " MB of memory on GPU for local_max_vals" << endl;
	cout << "Allocating " << (sizeof(*max_response_r) * r * c * z)/(double)(1024*1024) << " MB of memory on GPU for max_response_r" << endl;
	cout << "Allocating " << (sizeof(*max_response_c) * r * c * z)/(double)(1024*1024) << " MB of memory on GPU for max_response_c" << endl;
	cout << "Allocating " << (sizeof(*max_response_z) * r * c * z)/(double)(1024*1024) << " MB of memory on GPU for max_response_z" << endl;
	
	//Allocate memory on device
	errorcode = hipMalloc((void**) &dev_im_vals, r * c * z * sizeof(*im_vals));
	errorcode = hipMalloc((void**) &dev_local_max_vals, r * c * z * sizeof(*local_max_vals));
	errorcode = hipMalloc((void**) &dev_max_response_r, r * c * z * sizeof(*dev_max_response_r));
	errorcode = hipMalloc((void**) &dev_max_response_c, r * c * z * sizeof(*dev_max_response_c));
	errorcode = hipMalloc((void**) &dev_max_response_z, r * c * z * sizeof(*dev_max_response_z));

	//cout << errorcode << endl;

	//Copy host memory contents to device contents
	hipMemcpy(dev_im_vals, im_vals, r * c * z * sizeof(*im_vals), hipMemcpyHostToDevice);
	hipMemcpy(dev_local_max_vals, local_max_vals, r * c * z * sizeof(*local_max_vals), hipMemcpyHostToDevice);

	int device;
	hipDeviceProp_t device_prop;

	hipGetDevice(&device);
	hipGetDeviceProperties(&device_prop, device);
	
	int threadsPerBlock = device_prop.maxThreadsDim[0];
	int numBlocks = 16;
	
	//Run kernel repeatedly with offset since we cannot launch too many threads at once
	for (int k = 0; k < r * c * z; k+= numBlocks * threadsPerBlock) //Run kernel on 16K pixels at a time
	{
		InitialClusteringKernel<<< numBlocks , threadsPerBlock >>>(dev_im_vals, dev_local_max_vals, dev_max_response_r, dev_max_response_c, dev_max_response_z , r, c, z, scale_xy, scale_z, k);
	}
	
	//Copy device memory contents back to host memory
	hipMemcpy(max_response_r, dev_max_response_r, r * c * z * sizeof(*max_response_r), hipMemcpyDeviceToHost);
	hipMemcpy(max_response_c, dev_max_response_c, r * c * z * sizeof(*max_response_c), hipMemcpyDeviceToHost);
	hipMemcpy(max_response_z, dev_max_response_z, r * c * z * sizeof(*max_response_z), hipMemcpyDeviceToHost);

	cout << hipGetErrorString(hipGetLastError()) << endl;
	
	//Block until all precious commands are complete
	hipDeviceSynchronize();

	hipFree(dev_im_vals);
	hipFree(dev_local_max_vals);
	hipFree(max_response_r);
	hipFree(max_response_c);
	hipFree(max_response_z);
	
	hipDeviceReset();	
	
	cout << "CUDA done" << endl;
}